#include "hip/hip_runtime.h"
#include "epolar_chgpen.h"
#include "field_chgpen.h"
#include "induce_donly.h"
#include "launch.h"
#include "mod.uprior.h"
#include "tinker_rt.h"
#include "tool/cudalib.h"
#include "tool/io_print.h"
#include <tinker/detail/inform.hh>
#include <tinker/detail/polpcg.hh>
#include <tinker/detail/polpot.hh>
#include <tinker/detail/units.hh>


namespace tinker {
#define ITHREAD threadIdx.x + blockIdx.x* blockDim.x
#define STRIDE  blockDim.x* gridDim.x


__global__
void pcg_udir_donly(int n, const real* restrict polarity,
                    real (*restrict udir)[3], const real (*restrict field)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli = polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         udir[i][j] = poli * field[i][j];
      }
   }
}


__global__
void pcg_rsd2(int n, const real* restrict polarity_inv, //
              real (*restrict rsd)[3],                  //
              const real (*restrict udir)[3], const real (*restrict uind)[3],
              const real (*restrict field)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         rsd[i][j] = (udir[i][j] - uind[i][j]) * poli_inv + field[i][j];
   }
}


__global__
void pcg_rsd1(int n, const real* restrict polarity, real (*restrict rsd)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
      }
   }
}


__global__
void pcg_p4(int n, const real* restrict polarity_inv, real (*restrict vec)[3],
            const real (*restrict conj)[3], const real (*restrict field)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         vec[i][j] = poli_inv * conj[i][j] - field[i][j];
   }
}


__global__
void pcg_p5(int n, const real* restrict polarity, //
            const real* restrict ka,              //
            const real* restrict ksum, real (*restrict uind)[3],
            const real (*restrict conj)[3], real (*restrict rsd)[3],
            const real (*restrict vec)[3])
{
   real kaval = *ka;
   real a = *ksum / kaval;
   if (kaval == 0)
      a = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += a * conj[i][j];
         rsd[i][j] -= a * vec[i][j];
      }
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
      }
   }
}


__global__
void pcg_p6(int n, const real* restrict ksum, const real* restrict ksum1,
            real (*restrict conj)[3], real (*restrict zrsd)[3])
{
   real ksumval = *ksum;
   real b = *ksum1 / ksumval;
   if (ksumval == 0)
      b = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         conj[i][j] = zrsd[i][j] + b * conj[i][j];
   }
}


__global__
void pcg_peek1(int n, float pcgpeek, const real* restrict polarity,
               real (*restrict uind)[3], const real (*restrict rsd)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real term = pcgpeek * polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j)
         uind[i][j] += term * rsd[i][j];
   }
}


void induce_mutual_pcg2_cu(real (*uind)[3])
{
   auto* field = work01_;
   auto* rsd = work02_;
   auto* zrsd = work03_;
   auto* conj = work04_;
   auto* vec = work05_;


   const bool sparse_prec = polpcg::pcgprec;
   bool dirguess = polpcg::pcgguess;
   bool predict = polpred != UPred::NONE;
   if (predict and nualt < maxualt) {
      predict = false;
      dirguess = true;
   }


   // get the electrostatic field due to permanent multipoles
   dfield_chgpen(field);
   // direct induced dipoles
   launch_k1s(g::s0, n, pcg_udir_donly, n, polarity, udir, field);


   // initial induced dipole
   if (predict) {
      ulspred_sum2(uind);
   } else if (dirguess) {
      darray::copy(g::q0, n, uind, udir);
   } else {
      darray::zero(g::q0, n, uind);
   }


   // initial residual r(0)
   //
   // if use pcgguess, r(0) = E - (inv_alpha + Tu) alpha E
   //                       = E - E -Tu udir
   //                       = -Tu udir
   //
   // in general, r(0) = E - (inv_alpha + Tu) u(0)
   //                  = -Tu u(0) + E - inv_alpha u(0)
   //                  = -Tu u(0) + inv_alpha (udir - u(0))
   //
   // if do not use pcgguess, r(0) = E - T Zero = E
   if (predict) {
      ufield_chgpen(uind, field);
      launch_k1s(g::s0, n, pcg_rsd2, n, polarity_inv, rsd, udir, uind, field);
   } else if (dirguess) {
      ufield_chgpen(udir, rsd);
   } else {
      darray::copy(g::q0, n, rsd, field);
   }
   launch_k1s(g::s0, n, pcg_rsd1, n, polarity, rsd);


   // initial M r(0) and p(0)
   if (sparse_prec) {
      sparse_precond_build2();
      sparse_precond_apply2(rsd, zrsd);
   } else {
      diag_precond2(rsd, zrsd);
   }
   darray::copy(g::q0, n, conj, zrsd);


   // initial r(0) M r(0)
   real* sum = &((real*)dptr_buf)[0];
   darray::dot(g::q0, n, sum, rsd, zrsd);


   // conjugate gradient iteration of the mutual induced dipoles
   const bool debug = inform::debug;
   const int politer = polpot::politer;
   const real poleps = polpot::poleps;
   const real debye = units::debye;
   const real pcgpeek = polpcg::pcgpeek;
   const int maxiter = 100; // see also subroutine induce0a in induce.f


   bool done = false;
   int iter = 0;
   real eps = 100;
   real epsold;


   while (!done) {
      ++iter;


      // T p and p
      // vec = (inv_alpha + Tu) conj, field = -Tu conj
      // vec = inv_alpha * conj - field
      ufield_chgpen(conj, field);
      launch_k1s(g::s0, n, pcg_p4, n, polarity_inv, vec, conj, field);


      // a <- p T p
      real* a = &((real*)dptr_buf)[1];
      // a <- r M r / p T p; a = sum / a; ap = sump / ap
      darray::dot(g::q0, n, a, conj, vec);


      // u <- u + a p
      // r <- r - a T p
      launch_k1s(g::s0, n, pcg_p5, n, polarity, a, sum, uind, conj, rsd, vec);


      // calculate/update M r
      if (sparse_prec)
         sparse_precond_apply2(rsd, zrsd);
      else
         diag_precond2(rsd, zrsd);


      // b = sum1 / sum; bp = sump1 / sump
      real* sum1 = &((real*)dptr_buf)[2];
      darray::dot(g::q0, n, sum1, rsd, zrsd);


      // calculate/update p
      launch_k1s(g::s0, n, pcg_p6, n, sum, sum1, conj, zrsd);


      // copy sum1/p to sum/p
      darray::copy(g::q0, 2, sum, sum1);


      real* epsd = &((real*)dptr_buf)[3];
      darray::dot(g::q0, n, epsd, rsd, rsd);
      check_rt(hipMemcpyAsync((real*)pinned_buf, epsd, sizeof(real),
                               hipMemcpyDeviceToHost, g::s0));
      check_rt(hipStreamSynchronize(g::s0));
      epsold = eps;
      eps = ((real*)pinned_buf)[0];
      eps = debye * REAL_SQRT(eps / n);


      if (debug) {
         if (iter == 1) {
            print(stdout,
                  "\n Determination of SCF Induced Dipole Moments\n\n"
                  "    Iter    RMS Residual (Debye)\n\n");
         }
         print(stdout, " %8d       %-16.10f\n", iter, eps);
      }


      if (eps < poleps)
         done = true;
      if (eps > epsold)
         done = true;
      if (iter >= politer)
         done = true;


      // apply a "peek" iteration to the mutual induced dipoles
      if (done)
         launch_k1s(g::s0, n, pcg_peek1, n, pcgpeek, polarity, uind, rsd);
   }


   // print the results from the conjugate gradient iteration
   if (debug) {
      print(stdout,
            " Induced Dipoles :    Iterations %4d      RMS"
            " Residual %14.10f\n",
            iter, eps);
   }


   // terminate the calculation if dipoles failed to converge
   if (iter >= maxiter || eps > epsold) {
      t_prterr();
      TINKER_THROW("INDUCE  --  Warning, Induced Dipoles are not Converged");
   }
}
}
