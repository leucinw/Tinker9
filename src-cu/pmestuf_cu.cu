#include "hip/hip_runtime.h"
#include "add.h"
#include "box.h"
#include "empole.h"
#include "glob.spatial.h"
#include "launch.h"
#include "md.h"
#include "pmestuf.h"
#include "seq_bsplgen.h"
#include "tool/gpu_card.h"


namespace tinker {
// compute theta values on the fly
template <class T, int bsorder>
__global__
void grid_put_cu1(const real* restrict x, const real* restrict y,
                  const real* restrict z, int n, int nfft1, int nfft2,
                  int nfft3, const real* restrict ptr1,
                  const real* restrict ptr2, real* restrict qgrid,
                  real3 recip_a, real3 recip_b, real3 recip_c)
{
   real thetai1[4 * 5];
   real thetai2[4 * 5];
   real thetai3[4 * 5];
   __shared__ real sharedarray[5 * 5 * PME_BLOCKDIM];
   real* restrict array = &sharedarray[5 * 5 * threadIdx.x];


   MAYBE_UNUSED const real* pchg = ptr1;
   MAYBE_UNUSED const real(*fmp)[10] = (real(*)[10])ptr1;
   MAYBE_UNUSED const real(*fuind)[3] = (real(*)[3])ptr1;
   MAYBE_UNUSED const real(*fuinp)[3] = (real(*)[3])ptr2;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);


      if CONSTEXPR (eq<T, PCHG>() || eq<T, DISP>()) {
         real chgi = pchg[i];
         if (chgi == 0)
            continue;


         bsplgen<1, bsorder>(w1, thetai1, array);
         bsplgen<1, bsorder>(w2, thetai2, array);
         bsplgen<1, bsorder>(w3, thetai3, array);


         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz] * chgi;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy] * v0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real term = thetai1[4 * ix] * u0;
                  atomic_add(term, qgrid, 2 * index);
               }
            }
         }
      } // end if (PCHG or DISP)


      if CONSTEXPR (eq<T, MPOLE>()) {
         bsplgen<3, bsorder>(w1, thetai1, array);
         bsplgen<3, bsorder>(w2, thetai2, array);
         bsplgen<3, bsorder>(w3, thetai3, array);


         real fmpi0 = fmp[i][mpl_pme_0];
         real fmpix = fmp[i][mpl_pme_x];
         real fmpiy = fmp[i][mpl_pme_y];
         real fmpiz = fmp[i][mpl_pme_z];
         real fmpixx = fmp[i][mpl_pme_xx];
         real fmpiyy = fmp[i][mpl_pme_yy];
         real fmpizz = fmp[i][mpl_pme_zz];
         real fmpixy = fmp[i][mpl_pme_xy];
         real fmpixz = fmp[i][mpl_pme_xz];
         real fmpiyz = fmp[i][mpl_pme_yz];
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               // fmp: 0, x, y, z, xx, yy, zz, xy, xz, yz
               //      1, 2, 3, 4,  5,  6,  7,  8,  9, 10
               real term0 = fmpi0 * u0 * v0 + fmpiy * u1 * v0 +
                  fmpiz * u0 * v1 + fmpiyy * u2 * v0 + fmpizz * u0 * v2 +
                  fmpiyz * u1 * v1;
               real term1 =
                  fmpix * u0 * v0 + fmpixy * u1 * v0 + fmpixz * u0 * v1;
               real term2 = fmpixx * u0 * v0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  real t2 = thetai1[4 * ix + 2];
                  atomic_add(term0 * t0 + term1 * t1 + term2 * t2, qgrid,
                             2 * index);
               }
            } // end for (int iy)
         }
      } // end if (MPOLE)


      if CONSTEXPR (eq<T, UIND>()) {
         bsplgen<2, bsorder>(w1, thetai1, array);
         bsplgen<2, bsorder>(w2, thetai2, array);
         bsplgen<2, bsorder>(w3, thetai3, array);


         real fuindi0 = fuind[i][0];
         real fuindi1 = fuind[i][1];
         real fuindi2 = fuind[i][2];
         real fuinpi0 = fuinp[i][0];
         real fuinpi1 = fuinp[i][1];
         real fuinpi2 = fuinp[i][2];
         #pragma unroll
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            #pragma unroll
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real term01 = fuindi1 * u1 * v0 + fuindi2 * u0 * v1;
               real term11 = fuindi0 * u0 * v0;
               real term02 = fuinpi1 * u1 * v0 + fuinpi2 * u0 * v1;
               real term12 = fuinpi0 * u0 * v0;
               #pragma unroll
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  int index = xbase + ybase + zbase;
                  real t0 = thetai1[4 * ix];
                  real t1 = thetai1[4 * ix + 1];
                  atomic_add(term01 * t0 + term11 * t1, qgrid, 2 * index);
                  atomic_add(term02 * t0 + term12 * t1, qgrid, 2 * index + 1);
               }
            } // end for (int iy)
         }
      } // end if (UIND)
   }
}


// use pre-computed theta values
template <class T, int bsorder>
__global__
void grid_put_cu2(const int* restrict igrid, const real* restrict thetai1,
                  const real* restrict thetai2, const real* restrict thetai3,
                  const Spatial::SortedAtom* restrict sorted, int n,
                  int padded_n, int nfft1, int nfft2, int nfft3,
                  const real* restrict ptr1, const real* ptr2,
                  real* restrict qgrid)
{
   constexpr int bso2 = bsorder * bsorder;
   constexpr int bso3 = bsorder * bso2;
   for (int m = threadIdx.x + blockIdx.x * blockDim.x; m < n * bso3;
        m += blockDim.x * gridDim.x) {
      // m = i0 * bso3 + j;
      int i0 = m / bso3;
      int j = m - i0 * bso3;


      int i = sorted[i0].unsorted;
      int igrid1 = igrid[3 * i + 0];
      int igrid2 = igrid[3 * i + 1];
      int igrid3 = igrid[3 * i + 2];


      int iz = j / bso2;
      j -= iz * bso2;
      int iy = j / bsorder;
      int ix = j - (j / bsorder) * bsorder;


      real v0 = thetai3[(4 * iz + 0) * padded_n + i];
      real v1 = thetai3[(4 * iz + 1) * padded_n + i];
      int zbase = igrid3 + iz;
      zbase -= (zbase >= nfft3 ? nfft3 : 0);
      zbase *= (nfft1 * nfft2);


      real u0 = thetai2[(4 * iy + 0) * padded_n + i];
      real u1 = thetai2[(4 * iy + 1) * padded_n + i];
      int ybase = igrid2 + iy;
      ybase -= (ybase >= nfft2 ? nfft2 : 0);
      ybase *= nfft1;


      real t0 = thetai1[(4 * ix + 0) * padded_n + i];
      real t1 = thetai1[(4 * ix + 1) * padded_n + i];
      int xbase = igrid1 + ix;
      xbase -= (xbase >= nfft1 ? nfft1 : 0);
      int index = xbase + ybase + zbase;


      if CONSTEXPR (eq<T, MPOLE>()) {
         real v2 = thetai3[(4 * iz + 2) * padded_n + i];
         real u2 = thetai2[(4 * iy + 2) * padded_n + i];
         real t2 = thetai1[(4 * ix + 2) * padded_n + i];
         real fmpi0 = ptr1[i * 10 + mpl_pme_0];
         real fmpix = ptr1[i * 10 + mpl_pme_x];
         real fmpiy = ptr1[i * 10 + mpl_pme_y];
         real fmpiz = ptr1[i * 10 + mpl_pme_z];
         real fmpixx = ptr1[i * 10 + mpl_pme_xx];
         real fmpiyy = ptr1[i * 10 + mpl_pme_yy];
         real fmpizz = ptr1[i * 10 + mpl_pme_zz];
         real fmpixy = ptr1[i * 10 + mpl_pme_xy];
         real fmpixz = ptr1[i * 10 + mpl_pme_xz];
         real fmpiyz = ptr1[i * 10 + mpl_pme_yz];
         real term0 = fmpi0 * u0 * v0 + fmpiy * u1 * v0 + fmpiz * u0 * v1 +
            fmpiyy * u2 * v0 + fmpizz * u0 * v2 + fmpiyz * u1 * v1;
         real term1 = fmpix * u0 * v0 + fmpixy * u1 * v0 + fmpixz * u0 * v1;
         real term2 = fmpixx * u0 * v0;
         atomic_add(term0 * t0 + term1 * t1 + term2 * t2, qgrid, 2 * index);
      }


      if CONSTEXPR (eq<T, UIND>()) {
         real3 fd =
            make_real3(ptr1[3 * i + 0], ptr1[3 * i + 1], ptr1[3 * i + 2]);
         real3 fp =
            make_real3(ptr2[3 * i + 0], ptr2[3 * i + 1], ptr2[3 * i + 2]);
         real3 tuv = make_real3(t1 * u0 * v0, t0 * u1 * v0, t0 * u0 * v1);
         atomic_add(dot3(fd, tuv), qgrid, 2 * index);
         atomic_add(dot3(fp, tuv), qgrid, 2 * index + 1);
      }
   }
}


void grid_pchg_cu(PMEUnit pme_u, real* pchg)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   auto stream = g::s0;
   if (use_pme_stream)
      stream = g::spme;
   using type = std::remove_pointer<decltype(st.qgrid)>::type;
   check_rt(hipMemsetAsync(st.qgrid, 0, 2 * nt * sizeof(type), stream));
   if (st.bsorder == 5) {
      auto ker = grid_put_cu1<PCHG, 5>;
      launch_k2s(stream, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3, pchg,
                 nullptr, st.qgrid, recipa, recipb, recipc);
   } else if (st.bsorder == 4) {
      auto ker = grid_put_cu1<PCHG, 4>;
      launch_k2s(stream, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3, pchg,
                 nullptr, st.qgrid, recipa, recipb, recipc);
   }
}


void grid_disp_cu(PMEUnit pme_u, real* csix)
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   darray::zero(g::q0, 2 * nt, st.qgrid);
   auto ker = grid_put_cu1<DISP, 4>;
   launch_k2s(g::s0, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3, csix,
              nullptr, st.qgrid, recipa, recipb, recipc);
}


void grid_mpole_cu(PMEUnit pme_u, real (*fmp)[10])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   darray::zero(g::q0, 2 * nt, st.qgrid);
   if (TINKER_CU_THETA_ON_THE_FLY_GRID_MPOLE) {
      auto ker = grid_put_cu1<MPOLE, 5>;
      launch_k2s(g::s0, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
                 (const real*)fmp, nullptr, st.qgrid, recipa, recipb, recipc);
   } else {
      auto ker = grid_put_cu2<MPOLE, 5>;
      int npa = 5 * 5 * 5 * n;
      launch_k1s(g::s0, npa, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
                 mspatial_v2_unit->sorted, n, padded_n, n1, n2, n3,
                 (const real*)fmp, nullptr, st.qgrid);
   }
}


void grid_uind_cu(PMEUnit pme_u, real (*fuind)[3], real (*fuinp)[3])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;
   int nt = n1 * n2 * n3;


   darray::zero(g::q0, 2 * nt, st.qgrid);
   if (TINKER_CU_THETA_ON_THE_FLY_GRID_UIND) {
      auto ker = grid_put_cu1<UIND, 5>;
      launch_k2s(g::s0, PME_BLOCKDIM, n, ker, x, y, z, n, n1, n2, n3,
                 (const real*)fuind, (const real*)fuinp, st.qgrid, recipa,
                 recipb, recipc);
   } else {
      auto ker = grid_put_cu2<UIND, 5>;
      int npa = 5 * 5 * 5 * n;
      launch_k1s(g::s0, npa, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3,
                 mspatial_v2_unit->sorted, n, padded_n, n1, n2, n3,
                 (const real*)fuind, (const real*)fuinp, st.qgrid);
   }
}


template <int LEVEL, int bsorder>
__global__
void bspline_fill_cu1(int* restrict igrid, real* restrict thetai1,
                      real* restrict thetai2, real* restrict thetai3,
                      const real* restrict x, const real* restrict y,
                      const real* restrict z, int n, int padded_n, int nfft1,
                      int nfft2, int nfft3, real3 recip_a, real3 recip_b,
                      real3 recip_c)
{
   const int nfft4[3] = {nfft1, nfft2, nfft3};
   const real3 recip4[3] = {recip_a, recip_b, recip_c};
   real* const thetai[3] = {thetai1, thetai2, thetai3};
   real array[5 * 5];


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];
      int igridi[3];
      for (int j = 0; j < 3; ++j) {
         real w4 = xi * recip4[j].x + yi * recip4[j].y + zi * recip4[j].z;
         w4 = w4 + 0.5f - REAL_FLOOR(w4 + 0.5f);
         real fr4 = nfft4[j] * w4;
         int igrid4 = REAL_FLOOR(fr4);
         w4 = fr4 - igrid4;
         igrid4 = igrid4 - bsorder + 1;
         igrid4 += (igrid4 < 0 ? nfft4[j] : 0);
         // write output
         igridi[j] = igrid4;
         bsplgen2<LEVEL, bsorder>(w4, thetai[j], i, padded_n, array);
      }
      igrid[3 * i + 0] = igridi[0];
      igrid[3 * i + 1] = igridi[1];
      igrid[3 * i + 2] = igridi[2];
   }
}


void bspline_fill_cu(PMEUnit u, int level)
{
   auto& st = *u;
   if (level == 2) {
      auto ker = bspline_fill_cu1<2, 5>;
      launch_k1s(g::s0, n, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3, x,
                 y, z, n, padded_n, st.nfft1, st.nfft2, st.nfft3, recipa,
                 recipb, recipc);
   } else if (level == 3) {
      auto ker = bspline_fill_cu1<3, 5>;
      launch_k1s(g::s0, n, ker, st.igrid, st.thetai1, st.thetai2, st.thetai3, x,
                 y, z, n, padded_n, st.nfft1, st.nfft2, st.nfft3, recipa,
                 recipb, recipc);
   }
}


// compute theta values on the fly
template <class T, int bsorder>
__global__
void fphi_get_cu(int n, int nfft1, int nfft2, int nfft3, const real* restrict x,
                 const real* restrict y, const real* restrict z,
                 real* restrict opt1, real* restrict opt2, real* restrict opt3,
                 const real* restrict qgrid, real3 recip_a, real3 recip_b,
                 real3 recip_c)
{
   real thetai1[4 * 5];
   real thetai2[4 * 5];
   real thetai3[4 * 5];
   real array[5 * 5];


   MAYBE_UNUSED real(*restrict fphi)[20] = (real(*)[20])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi1)[10] = (real(*)[10])opt1;
   MAYBE_UNUSED real(*restrict fdip_phi2)[10] = (real(*)[10])opt2;
   MAYBE_UNUSED real(*restrict fdip_sum_phi)[20] = (real(*)[20])opt3;


   for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
        i += blockDim.x * gridDim.x) {
      real xi = x[i];
      real yi = y[i];
      real zi = z[i];


      real w1 = xi * recip_a.x + yi * recip_a.y + zi * recip_a.z;
      w1 = w1 + 0.5f - REAL_FLOOR(w1 + 0.5f);
      real fr1 = nfft1 * w1;
      int igrid1 = REAL_FLOOR(fr1);
      w1 = fr1 - igrid1;


      real w2 = xi * recip_b.x + yi * recip_b.y + zi * recip_b.z;
      w2 = w2 + 0.5f - REAL_FLOOR(w2 + 0.5f);
      real fr2 = nfft2 * w2;
      int igrid2 = REAL_FLOOR(fr2);
      w2 = fr2 - igrid2;


      real w3 = xi * recip_c.x + yi * recip_c.y + zi * recip_c.z;
      w3 = w3 + 0.5f - REAL_FLOOR(w3 + 0.5f);
      real fr3 = nfft3 * w3;
      int igrid3 = REAL_FLOOR(fr3);
      w3 = fr3 - igrid3;


      igrid1 = igrid1 - bsorder + 1;
      igrid2 = igrid2 - bsorder + 1;
      igrid3 = igrid3 - bsorder + 1;
      igrid1 += (igrid1 < 0 ? nfft1 : 0);
      igrid2 += (igrid2 < 0 ? nfft2 : 0);
      igrid3 += (igrid3 < 0 ? nfft3 : 0);

      if CONSTEXPR (eq<T, MPOLE>() || eq<T, UIND>() || eq<T, UIND2>()) {
         bsplgen<4, bsorder>(w1, thetai1, array);
         bsplgen<4, bsorder>(w2, thetai2, array);
         bsplgen<4, bsorder>(w3, thetai3, array);
      }


      if CONSTEXPR (eq<T, MPOLE>()) {
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0 = 0;
               real t1 = 0;
               real t2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq = qgrid[2 * (xbase + ybase + zbase)];
                  t0 += tq * thetai1[4 * ix];
                  t1 += tq * thetai1[4 * ix + 1];
                  t2 += tq * thetai1[4 * ix + 2];
                  t3 += tq * thetai1[4 * ix + 3];
               }
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         }
         fphi[i][0] = tuv000;
         fphi[i][1] = tuv100;
         fphi[i][2] = tuv010;
         fphi[i][3] = tuv001;
         fphi[i][4] = tuv200;
         fphi[i][5] = tuv020;
         fphi[i][6] = tuv002;
         fphi[i][7] = tuv110;
         fphi[i][8] = tuv101;
         fphi[i][9] = tuv011;
         fphi[i][10] = tuv300;
         fphi[i][11] = tuv030;
         fphi[i][12] = tuv003;
         fphi[i][13] = tuv210;
         fphi[i][14] = tuv201;
         fphi[i][15] = tuv120;
         fphi[i][16] = tuv021;
         fphi[i][17] = tuv102;
         fphi[i][18] = tuv012;
         fphi[i][19] = tuv111;
      }


      if CONSTEXPR (eq<T, UIND>()) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         real tuv000 = 0;
         real tuv001 = 0;
         real tuv010 = 0;
         real tuv100 = 0;
         real tuv200 = 0;
         real tuv020 = 0;
         real tuv002 = 0;
         real tuv110 = 0;
         real tuv101 = 0;
         real tuv011 = 0;
         real tuv300 = 0;
         real tuv030 = 0;
         real tuv003 = 0;
         real tuv210 = 0;
         real tuv201 = 0;
         real tuv120 = 0;
         real tuv021 = 0;
         real tuv102 = 0;
         real tuv012 = 0;
         real tuv111 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real v3 = thetai3[4 * iz + 3];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            real tu00 = 0;
            real tu10 = 0;
            real tu01 = 0;
            real tu20 = 0;
            real tu11 = 0;
            real tu02 = 0;
            real tu30 = 0;
            real tu21 = 0;
            real tu12 = 0;
            real tu03 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real u3 = thetai2[4 * iy + 3];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               real t3 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
                  t3 += (tq_1 + tq_2) * thetai1[4 * ix + 3];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
               real t0 = t0_1 + t0_2;
               real t1 = t1_1 + t1_2;
               real t2 = t2_1 + t2_2;
               tu00 += t0 * u0;
               tu10 += t1 * u0;
               tu01 += t0 * u1;
               tu20 += t2 * u0;
               tu11 += t1 * u1;
               tu02 += t0 * u2;
               tu30 += t3 * u0;
               tu21 += t2 * u1;
               tu12 += t1 * u2;
               tu03 += t0 * u3;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
            tuv000 += tu00 * v0;
            tuv100 += tu10 * v0;
            tuv010 += tu01 * v0;
            tuv001 += tu00 * v1;
            tuv200 += tu20 * v0;
            tuv020 += tu02 * v0;
            tuv002 += tu00 * v2;
            tuv110 += tu11 * v0;
            tuv101 += tu10 * v1;
            tuv011 += tu01 * v1;
            tuv300 += tu30 * v0;
            tuv030 += tu03 * v0;
            tuv003 += tu00 * v3;
            tuv210 += tu21 * v0;
            tuv201 += tu20 * v1;
            tuv120 += tu12 * v0;
            tuv021 += tu02 * v1;
            tuv102 += tu10 * v2;
            tuv012 += tu01 * v2;
            tuv111 += tu11 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
         fdip_sum_phi[i][0] = tuv000;
         fdip_sum_phi[i][1] = tuv100;
         fdip_sum_phi[i][2] = tuv010;
         fdip_sum_phi[i][3] = tuv001;
         fdip_sum_phi[i][4] = tuv200;
         fdip_sum_phi[i][5] = tuv020;
         fdip_sum_phi[i][6] = tuv002;
         fdip_sum_phi[i][7] = tuv110;
         fdip_sum_phi[i][8] = tuv101;
         fdip_sum_phi[i][9] = tuv011;
         fdip_sum_phi[i][10] = tuv300;
         fdip_sum_phi[i][11] = tuv030;
         fdip_sum_phi[i][12] = tuv003;
         fdip_sum_phi[i][13] = tuv210;
         fdip_sum_phi[i][14] = tuv201;
         fdip_sum_phi[i][15] = tuv120;
         fdip_sum_phi[i][16] = tuv021;
         fdip_sum_phi[i][17] = tuv102;
         fdip_sum_phi[i][18] = tuv012;
         fdip_sum_phi[i][19] = tuv111;
      }


      if CONSTEXPR (eq<T, UIND2>()) {
         real tuv100_1 = 0;
         real tuv010_1 = 0;
         real tuv001_1 = 0;
         real tuv200_1 = 0;
         real tuv020_1 = 0;
         real tuv002_1 = 0;
         real tuv110_1 = 0;
         real tuv101_1 = 0;
         real tuv011_1 = 0;
         real tuv100_2 = 0;
         real tuv010_2 = 0;
         real tuv001_2 = 0;
         real tuv200_2 = 0;
         real tuv020_2 = 0;
         real tuv002_2 = 0;
         real tuv110_2 = 0;
         real tuv101_2 = 0;
         real tuv011_2 = 0;
         for (int iz = 0; iz < bsorder; ++iz) {
            int zbase = igrid3 + iz;
            zbase -= (zbase >= nfft3 ? nfft3 : 0);
            zbase *= (nfft1 * nfft2);
            real v0 = thetai3[4 * iz];
            real v1 = thetai3[4 * iz + 1];
            real v2 = thetai3[4 * iz + 2];
            real tu00_1 = 0;
            real tu01_1 = 0;
            real tu10_1 = 0;
            real tu20_1 = 0;
            real tu11_1 = 0;
            real tu02_1 = 0;
            real tu00_2 = 0;
            real tu01_2 = 0;
            real tu10_2 = 0;
            real tu20_2 = 0;
            real tu11_2 = 0;
            real tu02_2 = 0;
            for (int iy = 0; iy < bsorder; ++iy) {
               int ybase = igrid2 + iy;
               ybase -= (ybase >= nfft2 ? nfft2 : 0);
               ybase *= nfft1;
               real u0 = thetai2[4 * iy];
               real u1 = thetai2[4 * iy + 1];
               real u2 = thetai2[4 * iy + 2];
               real t0_1 = 0;
               real t1_1 = 0;
               real t2_1 = 0;
               real t0_2 = 0;
               real t1_2 = 0;
               real t2_2 = 0;
               for (int ix = 0; ix < bsorder; ++ix) {
                  int xbase = igrid1 + ix;
                  xbase -= (xbase >= nfft1 ? nfft1 : 0);
                  real tq_1 = qgrid[2 * (xbase + ybase + zbase)];
                  real tq_2 = qgrid[2 * (xbase + ybase + zbase) + 1];
                  t0_1 += tq_1 * thetai1[4 * ix];
                  t1_1 += tq_1 * thetai1[4 * ix + 1];
                  t2_1 += tq_1 * thetai1[4 * ix + 2];
                  t0_2 += tq_2 * thetai1[4 * ix];
                  t1_2 += tq_2 * thetai1[4 * ix + 1];
                  t2_2 += tq_2 * thetai1[4 * ix + 2];
               }
               tu00_1 += t0_1 * u0;
               tu10_1 += t1_1 * u0;
               tu01_1 += t0_1 * u1;
               tu20_1 += t2_1 * u0;
               tu11_1 += t1_1 * u1;
               tu02_1 += t0_1 * u2;
               tu00_2 += t0_2 * u0;
               tu10_2 += t1_2 * u0;
               tu01_2 += t0_2 * u1;
               tu20_2 += t2_2 * u0;
               tu11_2 += t1_2 * u1;
               tu02_2 += t0_2 * u2;
            }
            tuv100_1 += tu10_1 * v0;
            tuv010_1 += tu01_1 * v0;
            tuv001_1 += tu00_1 * v1;
            tuv200_1 += tu20_1 * v0;
            tuv020_1 += tu02_1 * v0;
            tuv002_1 += tu00_1 * v2;
            tuv110_1 += tu11_1 * v0;
            tuv101_1 += tu10_1 * v1;
            tuv011_1 += tu01_1 * v1;
            tuv100_2 += tu10_2 * v0;
            tuv010_2 += tu01_2 * v0;
            tuv001_2 += tu00_2 * v1;
            tuv200_2 += tu20_2 * v0;
            tuv020_2 += tu02_2 * v0;
            tuv002_2 += tu00_2 * v2;
            tuv110_2 += tu11_2 * v0;
            tuv101_2 += tu10_2 * v1;
            tuv011_2 += tu01_2 * v1;
         } // end for (iz)
         fdip_phi1[i][0] = 0;
         fdip_phi1[i][1] = tuv100_1;
         fdip_phi1[i][2] = tuv010_1;
         fdip_phi1[i][3] = tuv001_1;
         fdip_phi1[i][4] = tuv200_1;
         fdip_phi1[i][5] = tuv020_1;
         fdip_phi1[i][6] = tuv002_1;
         fdip_phi1[i][7] = tuv110_1;
         fdip_phi1[i][8] = tuv101_1;
         fdip_phi1[i][9] = tuv011_1;
         fdip_phi2[i][0] = 0;
         fdip_phi2[i][1] = tuv100_2;
         fdip_phi2[i][2] = tuv010_2;
         fdip_phi2[i][3] = tuv001_2;
         fdip_phi2[i][4] = tuv200_2;
         fdip_phi2[i][5] = tuv020_2;
         fdip_phi2[i][6] = tuv002_2;
         fdip_phi2[i][7] = tuv110_2;
         fdip_phi2[i][8] = tuv101_2;
         fdip_phi2[i][9] = tuv011_2;
      }
   }
}


void fphi_mpole_cu(PMEUnit pme_u, real (*fphi)[20])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_get_cu<MPOLE, 5>;
   launch_k2s(g::s0, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z, (real*)fphi,
              nullptr, nullptr, st.qgrid, recipa, recipb, recipc);
}


void fphi_uind_cu(PMEUnit pme_u, real (*fdip_phi1)[10], real (*fdip_phi2)[10],
                  real (*fdip_sum_phi)[20])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_get_cu<UIND, 5>;
   launch_k2s(g::s0, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z,
              (real*)fdip_phi1, (real*)fdip_phi2, (real*)fdip_sum_phi, st.qgrid,
              recipa, recipb, recipc);
}


void fphi_uind2_cu(PMEUnit pme_u, real (*fdip_phi1)[10], real (*fdip_phi2)[10])
{
   auto& st = *pme_u;
   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   auto ker = fphi_get_cu<UIND2, 5>;
   launch_k2s(g::s0, PME_BLOCKDIM, n, ker, n, n1, n2, n3, x, y, z,
              (real*)fdip_phi1, (real*)fdip_phi2, nullptr, st.qgrid, recipa,
              recipb, recipc);
}


template <bool DO_E, bool DO_V>
__global__
void pme_conv_cu1(int nfft1, int nfft2, int nfft3, real (*restrict qgrid)[2],
                  const real* restrict bsmod1, const real* restrict bsmod2,
                  const real* restrict bsmod3, real f, real aewald,
                  TINKER_IMAGE_PARAMS, real box_volume,
                  energy_buffer restrict gpu_e, virial_buffer restrict gpu_vir)
{
   int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   int stride = blockDim.x * gridDim.x;


   int nff = nfft1 * nfft2;
   int ntot = nfft1 * nfft2 * nfft3;
   real pterm = pi / aewald;
   pterm *= pterm;


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec ectl;
   if CONSTEXPR (DO_E) {
      ectl = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz;
   if CONSTEXPR (DO_V) {
      vctlxx = 0;
      vctlyx = 0;
      vctlzx = 0;
      vctlyy = 0;
      vctlzy = 0;
      vctlzz = 0;
   }


   for (int i = ithread; i < ntot; i += stride) {
      if (i == 0) {
         qgrid[0][0] = 0;
         qgrid[0][1] = 0;
         continue;
      }


      int k3 = i / nff;
      int j = i - k3 * nff;
      int k2 = j / nfft1;
      int k1 = j - k2 * nfft1;


      int r1 = (k1 < (nfft1 + 1) / 2) ? k1 : (k1 - nfft1);
      int r2 = (k2 < (nfft2 + 1) / 2) ? k2 : (k2 - nfft2);
      int r3 = (k3 < (nfft3 + 1) / 2) ? k3 : (k3 - nfft3);


      real h1 = recipa.x * r1 + recipb.x * r2 + recipc.x * r3;
      real h2 = recipa.y * r1 + recipb.y * r2 + recipc.y * r3;
      real h3 = recipa.z * r1 + recipb.z * r2 + recipc.z * r3;
      real hsq = h1 * h1 + h2 * h2 + h3 * h3;


      real gridx = qgrid[i][0];
      real gridy = qgrid[i][1];
      real term = -pterm * hsq;
      real expterm = 0;
      if (term > -50) {
         real denom =
            hsq * pi * box_volume * bsmod1[k1] * bsmod2[k2] * bsmod3[k3];
         expterm = REAL_EXP(term) / denom;
         if (box_shape == UNBOUND_BOX)
            expterm *= (1 - REAL_COS(pi * lvec1.x * REAL_SQRT(hsq)));
         else if (box_shape == OCT_BOX)
            if ((k1 + k2 + k3) & 1)
               expterm = 0; // end if ((k1 + k2 + k3) % 2 != 0)


         if CONSTEXPR (DO_E || DO_V) {
            real struc2 = gridx * gridx + gridy * gridy;
            real eterm = 0.5f * f * expterm * struc2;
            if CONSTEXPR (DO_E) {
               ectl += cvt_to<ebuf_prec>(eterm);
            }
            if CONSTEXPR (DO_V) {
               real vterm = (2 / hsq) * (1 - term) * eterm;
               real vxx = (h1 * h1 * vterm - eterm);
               real vyx = h1 * h2 * vterm;
               real vzx = h1 * h3 * vterm;
               real vyy = (h2 * h2 * vterm - eterm);
               real vzy = h2 * h3 * vterm;
               real vzz = (h3 * h3 * vterm - eterm);
               vctlxx += cvt_to<vbuf_prec>(vxx);
               vctlyx += cvt_to<vbuf_prec>(vyx);
               vctlzx += cvt_to<vbuf_prec>(vzx);
               vctlyy += cvt_to<vbuf_prec>(vyy);
               vctlzy += cvt_to<vbuf_prec>(vzy);
               vctlzz += cvt_to<vbuf_prec>(vzz);
            }
         } // end if (e or v)
      }


      // complete the transformation of the PME grid
      qgrid[i][0] = gridx * expterm;
      qgrid[i][1] = gridy * expterm;
   }


   if CONSTEXPR (DO_E) {
      atomic_add(ectl, gpu_e, ithread);
   }
   if CONSTEXPR (DO_V) {
      atomic_add(vctlxx, vctlyx, vctlzx, vctlyy, vctlzy, vctlzz, gpu_vir,
                 ithread);
   }
}


template <bool DO_E, bool DO_V>
void pme_conv_cu2(PMEUnit pme_u, energy_buffer gpu_e, virial_buffer gpu_vir)
{
   auto& st = *pme_u;
   real(*restrict qgrid)[2] = reinterpret_cast<real(*)[2]>(st.qgrid);
   const real* bsmod1 = st.bsmod1;
   const real* bsmod2 = st.bsmod2;
   const real* bsmod3 = st.bsmod3;


   int n1 = st.nfft1;
   int n2 = st.nfft2;
   int n3 = st.nfft3;


   real f = electric / dielec;
   real aewald = st.aewald;
   real box_volume = volbox();


   auto ker = pme_conv_cu1<DO_E, DO_V>;
   auto stream = use_pme_stream ? g::spme : g::s0;
   int ngrid = get_grid_size(BLOCK_DIM);
   ker<<<ngrid, BLOCK_DIM, 0, stream>>>(n1, n2, n3, qgrid, bsmod1, bsmod2,
                                        bsmod3, f, aewald, TINKER_IMAGE_ARGS,
                                        box_volume, gpu_e, gpu_vir);
}


void pme_conv_cu(PMEUnit pme_u, energy_buffer gpu_e, virial_buffer gpu_vir)
{
   if (gpu_vir == nullptr) {
      if (gpu_e == nullptr) {
         pme_conv_cu2<false, false>(pme_u, nullptr, nullptr);
      } else {
         pme_conv_cu2<true, false>(pme_u, gpu_e, nullptr);
      }
   } else {
      if (gpu_e == nullptr) {
         pme_conv_cu2<false, true>(pme_u, nullptr, gpu_vir);
      } else {
         pme_conv_cu2<true, true>(pme_u, gpu_e, gpu_vir);
      }
   }
}
}
