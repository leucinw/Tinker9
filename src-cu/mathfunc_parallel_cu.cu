#include "hip/hip_runtime.h"
#include "mathfunc_parallel_cu.h"
#include "syntax/cu/reduce.h"
#include "tool/cudalib.h"
#include "tool/deduce_ptr.h"
#include "tool/error.h"
#include "tool/gpu_card.h"
#include <cassert>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <numeric>


namespace tinker {
namespace {
template <class T, class Op>
void reduce_to_dptr(const T* a, size_t nelem, hipStream_t st)
{
   T* dptr = (T*)dptr_buf;
   int grid_siz1 = get_grid_size(BLOCK_DIM);
   int grid_siz2 = (nelem + BLOCK_DIM - 1) / BLOCK_DIM;
   int grid_size = std::min(grid_siz1, grid_siz2);
   reduce<T, BLOCK_DIM, Op><<<grid_size, BLOCK_DIM, 0, st>>>(dptr, a, nelem);
   reduce<T, BLOCK_DIM, Op><<<1, BLOCK_DIM, 0, st>>>(dptr, dptr, grid_size);
}


template <class T, class Op>
T reduce_general(const T* a, size_t nelem, int queue)
{
   hipStream_t st = queue == g::q1 ? g::s1 : g::s0;
   T* dptr = (T*)dptr_buf;
   T* hptr = (T*)pinned_buf;
   reduce_to_dptr<T, Op>(a, nelem, st);
   check_rt(hipMemcpyAsync(hptr, dptr, sizeof(T), hipMemcpyDeviceToHost, st));
   // always wait
   check_rt(hipStreamSynchronize(st));
   return *hptr;
}
}


template <class T>
T reduce_sum_cu(const T* a, size_t nelem, int queue)
{
   return reduce_general<T, OpPlus<T>>(a, nelem, queue);
}
template int reduce_sum_cu(const int*, size_t, int);
template float reduce_sum_cu(const float*, size_t, int);
template double reduce_sum_cu(const double*, size_t, int);
template unsigned long long reduce_sum_cu(const unsigned long long*, size_t,
                                          int);


template <class HT, size_t HN, class DPTR>
void reduce_sum2_cu(HT (&restrict h_ans)[HN], DPTR restrict a, size_t nelem,
                    int queue)
{
   typedef typename deduce_ptr<DPTR>::type CONST_DT;
   typedef typename std::remove_const<CONST_DT>::type T;
   static_assert(std::is_same<HT, T>::value, "");
   constexpr size_t N = deduce_ptr<DPTR>::n;
   static_assert(HN <= N, "");

   hipStream_t st = queue == g::q1 ? g::s1 : g::s0;
   T(*dptr)[HN] = (T(*)[HN])dptr_buf;
   T* hptr = (T*)pinned_buf;
   int grid_siz1 = get_grid_size(BLOCK_DIM);
   grid_siz1 = grid_siz1 / HN; // limited by the output buffer
   int grid_siz2 = (nelem + BLOCK_DIM - 1) / BLOCK_DIM;
   int grid_size = std::min(grid_siz1, grid_siz2);
   reduce2<T, BLOCK_DIM, HN, N, OpPlus<T>>
      <<<grid_size, BLOCK_DIM, 0, st>>>(dptr, a, nelem);
   reduce2<T, BLOCK_DIM, HN, HN, OpPlus<T>>
      <<<1, BLOCK_DIM, 0, st>>>(dptr, dptr, grid_size);
   check_rt(hipMemcpyAsync(hptr, (T*)dptr, HN * sizeof(HT),
                            hipMemcpyDeviceToHost, st));
   // always wait
   check_rt(hipStreamSynchronize(st));
   #pragma unroll
   for (size_t j = 0; j < HN; ++j)
      h_ans[j] = hptr[j];
}
template void reduce_sum2_cu(float (&)[6], float (*)[8], size_t, int);
template void reduce_sum2_cu(double (&)[6], double (*)[8], size_t, int);
template void reduce_sum2_cu(unsigned long long (&)[6],
                             unsigned long long (*)[8], size_t, int);


template <class T>
void reduce_sum_on_device_cu(T* dp_ans, const T* a, size_t nelem, int queue)
{
   hipStream_t st = queue == g::q1 ? g::s1 : g::s0;
   T* dptr = (T*)dptr_buf;
   using Op = OpPlus<T>;


   int grid_siz1 = get_grid_size(BLOCK_DIM);
   int grid_siz2 = (nelem + BLOCK_DIM - 1) / BLOCK_DIM;
   int grid_size = std::min(grid_siz1, grid_siz2);
   reduce<T, BLOCK_DIM, Op><<<grid_size, BLOCK_DIM, 0, st>>>(dptr, a, nelem);
   reduce<T, BLOCK_DIM, Op><<<1, BLOCK_DIM, 0, st>>>(dp_ans, dptr, grid_size);
}
template void reduce_sum_on_device_cu(int*, const int*, size_t, int);
template void reduce_sum_on_device_cu(float*, const float*, size_t, int);
template void reduce_sum_on_device_cu(double*, const double*, size_t, int);
template void reduce_sum_on_device_cu(unsigned long long*,
                                      const unsigned long long*, size_t, int);


template <class HT, size_t HN, class DPTR>
void reduce_sum2_on_device_cu(HT (&dref)[HN], DPTR v, size_t nelem, int queue)
{
   typedef typename deduce_ptr<DPTR>::type CONST_DT;
   typedef typename std::remove_const<CONST_DT>::type T;
   static_assert(std::is_same<HT, T>::value, "");
   constexpr size_t N = deduce_ptr<DPTR>::n;
   static_assert(HN <= N, "");

   hipStream_t st = queue == g::q1 ? g::s1 : g::s0;
   T(*dptr)[HN] = (T(*)[HN])dptr_buf;
   T(*dpt2)[HN] = (T(*)[HN])dref;
   int grid_siz1 = get_grid_size(BLOCK_DIM);
   grid_siz1 = grid_siz1 / HN; // limited by the output buffer
   int grid_siz2 = (nelem + BLOCK_DIM - 1) / BLOCK_DIM;
   int grid_size = std::min(grid_siz1, grid_siz2);
   reduce2<T, BLOCK_DIM, HN, N, OpPlus<T>>
      <<<grid_size, BLOCK_DIM, 0, st>>>(dptr, v, nelem);
   reduce2<T, BLOCK_DIM, HN, HN, OpPlus<T>>
      <<<1, BLOCK_DIM, 0, st>>>(dpt2, dptr, grid_size);
}
template void reduce_sum2_on_device_cu(float (&)[6], float (*)[8], size_t, int);
template void reduce_sum2_on_device_cu(double (&)[6], double (*)[8], size_t,
                                       int);
template void reduce_sum2_on_device_cu(unsigned long long (&)[6],
                                       unsigned long long (*)[8], size_t, int);


template <>
void dotprod_cu<float>(float* ans, const float* a, const float* b, size_t nelem,
                       int queue)
{
   bool dq = queue == g::q1;
   hipblasHandle_t hd = (dq ? g::h1 : g::h0);
   check_rt(hipblasSdot(hd, nelem, a, 1, b, 1, ans));
}


template <>
void dotprod_cu<double>(double* ans, const double* a, const double* b,
                        size_t nelem, int queue)
{
   bool dq = queue == g::q1;
   hipblasHandle_t hd = (dq ? g::h1 : g::h0);
   check_rt(hipblasDdot(hd, nelem, a, 1, b, 1, ans));
}


// cublas gemm does not run as fast here prior to cuda 10.1.
// Old code:
//
// #if CUDART_VERSION >= 10100 // >= 10.1
//    float alpha = 1, beta = 0;
//    check_rt(hipblasSgemm(hd, HIPBLAS_OP_N, HIPBLAS_OP_T, 1, 1, nelem, //
//                         &alpha, a, 1, b, 1,                        //
//                         &beta, ans, 1));
// #else
//    check_rt(hipblasSdot(hd, nelem, a, 1, b, 1, ans));
// #endif
}
