#include "hip/hip_runtime.h"
#include "add.h"
#include "energy.h"
#include "glob.group.h"
#include "glob.molecule.h"
#include "image.h"
#include "launch.h"
#include "md.h"
#include "potent.h"
#include "seq_angle.h"
#include "seq_angtor.h"
#include "seq_bond.h"
#include "seq_geom.h"
#include "seq_improp.h"
#include "seq_imptor.h"
#include "seq_opbend.h"
#include "seq_pitors.h"
#include "seq_strbnd.h"
#include "seq_strtor.h"
#include "seq_torsion.h"
#include "seq_tortor.h"
#include "seq_urey.h"
#include "tool/gpu_card.h"
#include "tool/host_zero.h"


namespace tinker {
template <class Ver, bool rc_a>
__global__
void evalence_cu1(
   // ebond
   energy_buffer restrict eb, virial_buffer restrict vir_eb,
   grad_prec* restrict debx, grad_prec* restrict deby, grad_prec* restrict debz,

   ebond_t bndtyp, real bndunit, int nbond, const int (*restrict ibnd)[2],
   const real* restrict bl, const real* restrict bk, real cbnd, real qbnd,

   // eangle
   energy_buffer restrict ea, virial_buffer restrict vir_ea,
   grad_prec* restrict deax, grad_prec* restrict deay, grad_prec* restrict deaz,

   const eangle_t* restrict angtyp, real angunit, int nangle,
   const int (*restrict iang)[4], const real* restrict anat,
   const real* restrict ak, const real* restrict afld,

   real cang, real qang, real pang, real sang,

   // estrbnd
   energy_buffer restrict eba, virial_buffer restrict vir_eba,
   grad_prec* restrict debax, grad_prec* restrict debay,
   grad_prec* restrict debaz,

   real stbnunit, int nstrbnd, const int (*restrict isb)[3],
   const real (*restrict sbk)[2],

   // eurey
   energy_buffer restrict eub, virial_buffer restrict vir_eub,
   grad_prec* restrict deubx, grad_prec* restrict deuby,
   grad_prec* restrict deubz,

   real ureyunit, int nurey, const int (*restrict iury)[3],
   const real* restrict uk, const real* restrict ul, real cury, real qury,

   // eopbend
   energy_buffer restrict eopb, virial_buffer restrict vir_eopb,
   grad_prec* restrict deopbx, grad_prec* restrict deopby,
   grad_prec* restrict deopbz,

   eopbend_t opbtyp, real opbunit, int nopbend, const int* restrict iopb,
   const real* restrict opbk, real copb, real qopb, real popb, real sopb,

   // eimprop
   energy_buffer restrict eid, virial_buffer restrict vir_eid,
   grad_prec* restrict deidx, grad_prec* restrict deidy,
   grad_prec* restrict deidz,

   real idihunit, int niprop, const int (*restrict iiprop)[4],
   const real* restrict kprop, const real* restrict vprop,

   // eimptor
   energy_buffer restrict eit, virial_buffer restrict vir_eit,
   grad_prec* restrict deitx, grad_prec* restrict deity,
   grad_prec* restrict deitz,

   real itorunit, int nitors, const int (*restrict iitors)[4],
   const real (*restrict itors1)[4], const real (*restrict itors2)[4],
   const real (*restrict itors3)[4],

   // etors
   energy_buffer restrict et, virial_buffer restrict vir_et,
   grad_prec* restrict detx, grad_prec* restrict dety, grad_prec* restrict detz,

   real torsunit, int ntors, const int (*restrict itors)[4],
   const real (*restrict tors1)[4], const real (*restrict tors2)[4],
   const real (*restrict tors3)[4], const real (*restrict tors4)[4],
   const real (*restrict tors5)[4], const real (*restrict tors6)[4],

   // epitors
   energy_buffer restrict ept, virial_buffer restrict vir_ept,
   grad_prec* restrict deptx, grad_prec* restrict depty,
   grad_prec* restrict deptz,

   real ptorunit, int npitors, const int (*restrict ipit)[6],
   const real* restrict kpit,

   // estrtor
   energy_buffer restrict ebt, virial_buffer restrict vir_ebt,
   grad_prec* restrict debtx, grad_prec* restrict debty,
   grad_prec* restrict debtz,

   real storunit, int nstrtor, const int (*restrict ist)[4],
   const real (*restrict kst)[9],

   // eangtor
   energy_buffer restrict eat, virial_buffer restrict vir_eat,
   grad_prec* restrict deatx, grad_prec* restrict deaty,
   grad_prec* restrict deatz,

   real atorunit, int nangtor, const int (*restrict iat)[3],
   const real (*restrict kant)[6],

   // etortor
   energy_buffer restrict ett, virial_buffer restrict vir_ett,
   grad_prec* restrict dettx, grad_prec* restrict detty,
   grad_prec* restrict dettz,

   real ttorunit, int ntortor, const int (*restrict itt)[3],
   const int (*restrict ibitor)[5], const int* restrict chkttor_ia_,

   const int* restrict tnx, const int* restrict tny,
   const real (*restrict ttx)[ktrtor::maxtgrd],
   const real (*restrict tty)[ktrtor::maxtgrd],
   const real (*restrict tbf)[ktrtor::maxtgrd2],
   const real (*restrict tbx)[ktrtor::maxtgrd2],
   const real (*restrict tby)[ktrtor::maxtgrd2],
   const real (*restrict tbxy)[ktrtor::maxtgrd2],

   // egeom
   energy_buffer restrict eg, virial_buffer restrict vir_eg,
   grad_prec* restrict degx, grad_prec* restrict degy, grad_prec* restrict degz,

   int npfix, const int* restrict ipfix, const int (*restrict kpfix)[3],
   const real* restrict xpfix, const real* restrict ypfix,
   const real* restrict zpfix, const real (*restrict pfix)[2],

   int ngfix, const int (*restrict igfix)[2], const real (*restrict gfix)[3],

   int ndfix, const int (*restrict idfix)[2], const real (*restrict dfix)[3],

   int nafix, const int (*restrict iafix)[3], const real (*restrict afix)[3],

   int ntfix, const int (*restrict itfix)[4], const real (*restrict tfix)[3],

   // total
   energy_buffer restrict ebuf, virial_buffer restrict vbuf,

   // other
   const real* restrict x, const real* restrict y, const real* restrict z,
   const mass_prec* restrict mass, const int* restrict molec,
   const int (*restrict igrp)[2], const int* restrict kgrp,
   const mass_prec* restrict grpmass, TINKER_IMAGE_PARAMS)
{
   constexpr bool do_e = Ver::e;
   constexpr bool do_v = Ver::v;
   const int ithread = threadIdx.x + blockIdx.x * blockDim.x;
   const int stride = blockDim.x * gridDim.x;


   using ebuf_prec = energy_buffer_traits::type;
   ebuf_prec e0b;   // ebond
   ebuf_prec e0a;   // eangle
   ebuf_prec e0ba;  // estrbnd
   ebuf_prec e0ub;  // eurey
   ebuf_prec e0opb; // eopbend
   ebuf_prec e0id;  // eimprop
   ebuf_prec e0it;  // eimptor
   ebuf_prec e0t;   // etors
   ebuf_prec e0pt;  // epitors
   ebuf_prec e0bt;  // estrtor
   ebuf_prec e0at;  // eangtor
   ebuf_prec e0tt;  // etortor
   ebuf_prec e0g;   // egeom
   if CONSTEXPR (do_e) {
      e0b = 0;
      e0a = 0;
      e0ba = 0;
      e0ub = 0;
      e0opb = 0;
      e0id = 0;
      e0it = 0;
      e0t = 0;
      e0pt = 0;
      e0bt = 0;
      e0at = 0;
      e0tt = 0;
      e0g = 0;
   }
   using vbuf_prec = virial_buffer_traits::type;
   vbuf_prec v0bxx, v0byx, v0bzx, v0byy, v0bzy, v0bzz;             // ebond
   vbuf_prec v0axx, v0ayx, v0azx, v0ayy, v0azy, v0azz;             // eangle
   vbuf_prec v0baxx, v0bayx, v0bazx, v0bayy, v0bazy, v0bazz;       // estrbnd
   vbuf_prec v0ubxx, v0ubyx, v0ubzx, v0ubyy, v0ubzy, v0ubzz;       // eurey
   vbuf_prec v0opbxx, v0opbyx, v0opbzx, v0opbyy, v0opbzy, v0opbzz; // eopbend
   vbuf_prec v0idxx, v0idyx, v0idzx, v0idyy, v0idzy, v0idzz;       // eimprop
   vbuf_prec v0itxx, v0ityx, v0itzx, v0ityy, v0itzy, v0itzz;       // eimptor
   vbuf_prec v0txx, v0tyx, v0tzx, v0tyy, v0tzy, v0tzz;             // etors
   vbuf_prec v0ptxx, v0ptyx, v0ptzx, v0ptyy, v0ptzy, v0ptzz;       // epitors
   vbuf_prec v0btxx, v0btyx, v0btzx, v0btyy, v0btzy, v0btzz;       // estrtor
   vbuf_prec v0atxx, v0atyx, v0atzx, v0atyy, v0atzy, v0atzz;       // eangtor
   vbuf_prec v0ttxx, v0ttyx, v0ttzx, v0ttyy, v0ttzy, v0ttzz;       // etors
   vbuf_prec v0gxx, v0gyx, v0gzx, v0gyy, v0gzy, v0gzz;             // egeom
   if CONSTEXPR (do_v) {
      v0bxx = 0, v0byx = 0, v0bzx = 0, v0byy = 0, v0bzy = 0, v0bzz = 0;
      v0axx = 0, v0ayx = 0, v0azx = 0, v0ayy = 0, v0azy = 0, v0azz = 0;
      v0baxx = 0, v0bayx = 0, v0bazx = 0, v0bayy = 0, v0bazy = 0, v0bazz = 0;
      v0ubxx = 0, v0ubyx = 0, v0ubzx = 0, v0ubyy = 0, v0ubzy = 0, v0ubzz = 0;
      v0opbxx = 0, v0opbyx = 0, v0opbzx = 0;
      v0opbyy = 0, v0opbzy = 0, v0opbzz = 0;
      v0idxx = 0, v0idyx = 0, v0idzx = 0, v0idyy = 0, v0idzy = 0, v0idzz = 0;
      v0itxx = 0, v0ityx = 0, v0itzx = 0, v0ityy = 0, v0itzy = 0, v0itzz = 0;
      v0txx = 0, v0tyx = 0, v0tzx = 0, v0tyy = 0, v0tzy = 0, v0tzz = 0;
      v0ptxx = 0, v0ptyx = 0, v0ptzx = 0, v0ptyy = 0, v0ptzy = 0, v0ptzz = 0;
      v0btxx = 0, v0btyx = 0, v0btzx = 0, v0btyy = 0, v0btzy = 0, v0btzz = 0;
      v0atxx = 0, v0atyx = 0, v0atzx = 0, v0atyy = 0, v0atzy = 0, v0atzz = 0;
      v0ttxx = 0, v0ttyx = 0, v0ttzx = 0, v0ttyy = 0, v0ttzy = 0, v0ttzz = 0;
      v0gxx = 0, v0gyx = 0, v0gzx = 0, v0gyy = 0, v0gzy = 0, v0gzz = 0;
   }


   // ebond
   for (int i = ithread; i < nbond; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_bond<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                   debx, deby, debz,

                   bndtyp, bndunit, i, ibnd, bl, bk, cbnd, qbnd,

                   x, y, z);
      if CONSTEXPR (do_e) {
         e0b += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0bxx += cvt_to<vbuf_prec>(vxx);
         v0byx += cvt_to<vbuf_prec>(vyx);
         v0bzx += cvt_to<vbuf_prec>(vzx);
         v0byy += cvt_to<vbuf_prec>(vyy);
         v0bzy += cvt_to<vbuf_prec>(vzy);
         v0bzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nbond > 0)
         atomic_add(e0b, eb, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nbond > 0)
         atomic_add(v0bxx, v0byx, v0bzx, v0byy, v0bzy, v0bzz, vir_eb, ithread);
   }


   // eangle
   for (int i = ithread; i < nangle; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_angle<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                    deax, deay, deaz,

                    angtyp, angunit, i, iang, anat, ak, afld,

                    cang, qang, pang, sang,

                    x, y, z);
      if CONSTEXPR (do_e) {
         e0a += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0axx += cvt_to<vbuf_prec>(vxx);
         v0ayx += cvt_to<vbuf_prec>(vyx);
         v0azx += cvt_to<vbuf_prec>(vzx);
         v0ayy += cvt_to<vbuf_prec>(vyy);
         v0azy += cvt_to<vbuf_prec>(vzy);
         v0azz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nangle > 0)
         atomic_add(e0a, ea, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nangle > 0)
         atomic_add(v0axx, v0ayx, v0azx, v0ayy, v0azy, v0azz, vir_ea, ithread);
   }


   // estrbnd
   for (int i = ithread; i < nstrbnd; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_strbnd<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     debax, debay, debaz,

                     stbnunit, i, isb, sbk, bl, iang, anat,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0ba += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0baxx += cvt_to<vbuf_prec>(vxx);
         v0bayx += cvt_to<vbuf_prec>(vyx);
         v0bazx += cvt_to<vbuf_prec>(vzx);
         v0bayy += cvt_to<vbuf_prec>(vyy);
         v0bazy += cvt_to<vbuf_prec>(vzy);
         v0bazz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nstrbnd > 0)
         atomic_add(e0ba, eba, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nstrbnd > 0)
         atomic_add(v0baxx, v0bayx, v0bazx, v0bayy, v0bazy, v0bazz, vir_eba,
                    ithread);
   }


   // eurey
   for (int i = ithread; i < nurey; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_urey<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                   deubx, deuby, deubz,

                   ureyunit, i, iury, uk, ul, cury, qury,

                   x, y, z);
      if CONSTEXPR (do_e) {
         e0ub += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0ubxx += cvt_to<vbuf_prec>(vxx);
         v0ubyx += cvt_to<vbuf_prec>(vyx);
         v0ubzx += cvt_to<vbuf_prec>(vzx);
         v0ubyy += cvt_to<vbuf_prec>(vyy);
         v0ubzy += cvt_to<vbuf_prec>(vzy);
         v0ubzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nurey > 0)
         atomic_add(e0ub, eub, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nurey > 0)
         atomic_add(v0ubxx, v0ubyx, v0ubzx, v0ubyy, v0ubzy, v0ubzz, vir_eub,
                    ithread);
   }


   // eopbend
   for (int i = ithread; i < nopbend; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_opbend<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     deopbx, deopby, deopbz,

                     opbtyp, opbunit, i, iopb, opbk, iang, copb, qopb, popb,
                     sopb,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0opb += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0opbxx += cvt_to<vbuf_prec>(vxx);
         v0opbyx += cvt_to<vbuf_prec>(vyx);
         v0opbzx += cvt_to<vbuf_prec>(vzx);
         v0opbyy += cvt_to<vbuf_prec>(vyy);
         v0opbzy += cvt_to<vbuf_prec>(vzy);
         v0opbzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nopbend > 0)
         atomic_add(e0opb, eopb, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nopbend > 0)
         atomic_add(v0opbxx, v0opbyx, v0opbzx, v0opbyy, v0opbzy, v0opbzz,
                    vir_eopb, ithread);
   }


   // eimprop
   for (int i = ithread; i < niprop; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_improp<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     deidx, deidy, deidz,

                     idihunit, i, iiprop, kprop, vprop,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0id += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0idxx += cvt_to<vbuf_prec>(vxx);
         v0idyx += cvt_to<vbuf_prec>(vyx);
         v0idzx += cvt_to<vbuf_prec>(vzx);
         v0idyy += cvt_to<vbuf_prec>(vyy);
         v0idzy += cvt_to<vbuf_prec>(vzy);
         v0idzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (niprop > 0)
         atomic_add(e0id, eid, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (niprop > 0)
         atomic_add(v0idxx, v0idyx, v0idzx, v0idyy, v0idzy, v0idzz, vir_eid,
                    ithread);
   }


   // eimptor
   for (int i = ithread; i < nitors; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_imptor<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     deitx, deity, deitz,

                     itorunit, i, iitors, itors1, itors2, itors3,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0it += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0itxx += cvt_to<vbuf_prec>(vxx);
         v0ityx += cvt_to<vbuf_prec>(vyx);
         v0itzx += cvt_to<vbuf_prec>(vzx);
         v0ityy += cvt_to<vbuf_prec>(vyy);
         v0itzy += cvt_to<vbuf_prec>(vzy);
         v0itzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nitors > 0)
         atomic_add(e0it, eit, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nitors > 0)
         atomic_add(v0itxx, v0ityx, v0itzx, v0ityy, v0itzy, v0itzz, vir_eit,
                    ithread);
   }


   // etors
   for (int i = ithread; i < ntors; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_tors<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                   detx, dety, detz,

                   torsunit, i, itors,

                   tors1, tors2, tors3, tors4, tors5, tors6, x, y, z);
      if CONSTEXPR (do_e) {
         e0t += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0txx += cvt_to<vbuf_prec>(vxx);
         v0tyx += cvt_to<vbuf_prec>(vyx);
         v0tzx += cvt_to<vbuf_prec>(vzx);
         v0tyy += cvt_to<vbuf_prec>(vyy);
         v0tzy += cvt_to<vbuf_prec>(vzy);
         v0tzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (ntors > 0)
         atomic_add(e0t, et, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (ntors > 0)
         atomic_add(v0txx, v0tyx, v0tzx, v0tyy, v0tzy, v0tzz, vir_et, ithread);
   }


   // epitors
   for (int i = ithread; i < npitors; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_pitors<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     deptx, depty, deptz,

                     ptorunit, i, ipit, kpit, x, y, z);
      if CONSTEXPR (do_e) {
         e0pt += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0ptxx += cvt_to<vbuf_prec>(vxx);
         v0ptyx += cvt_to<vbuf_prec>(vyx);
         v0ptzx += cvt_to<vbuf_prec>(vzx);
         v0ptyy += cvt_to<vbuf_prec>(vyy);
         v0ptzy += cvt_to<vbuf_prec>(vzy);
         v0ptzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (npitors > 0)
         atomic_add(e0pt, ept, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (npitors > 0)
         atomic_add(v0ptxx, v0ptyx, v0ptzx, v0ptyy, v0ptzy, v0ptzz, vir_ept,
                    ithread);
   }

   // estrtor
   for (int i = ithread; i < nstrtor; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_strtor<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz, debtx, debty, debtz,

                     storunit, i, ist, kst,

                     bl, itors, tors1, tors2, tors3,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0bt += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0btxx += cvt_to<vbuf_prec>(vxx);
         v0btyx += cvt_to<vbuf_prec>(vyx);
         v0btzx += cvt_to<vbuf_prec>(vzx);
         v0btyy += cvt_to<vbuf_prec>(vyy);
         v0btzy += cvt_to<vbuf_prec>(vzy);
         v0btzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nstrtor > 0)
         atomic_add(e0bt, ebt, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nstrtor > 0)
         atomic_add(v0btxx, v0btyx, v0btzx, v0btyy, v0btzy, v0btzz, vir_ebt,
                    ithread);
   }

   // eangtor
   for (int i = ithread; i < nangtor; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_angtor<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz, deatx, deaty, deatz,

                     atorunit, i, iat, kant,

                     anat, itors, tors1, tors2, tors3,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0at += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0atxx += cvt_to<vbuf_prec>(vxx);
         v0atyx += cvt_to<vbuf_prec>(vyx);
         v0atzx += cvt_to<vbuf_prec>(vzx);
         v0atyy += cvt_to<vbuf_prec>(vyy);
         v0atzy += cvt_to<vbuf_prec>(vzy);
         v0atzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (nangtor > 0)
         atomic_add(e0at, eat, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (nangtor > 0)
         atomic_add(v0atxx, v0atyx, v0atzx, v0atyy, v0atzy, v0atzz, vir_eat,
                    ithread);
   }

   // etortor
   for (int i = ithread; i < ntortor; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_tortor<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                     dettx, detty, dettz,

                     ttorunit, i, itt, ibitor, chkttor_ia_,

                     tnx, tny, ttx, tty, tbf, tbx, tby, tbxy,

                     x, y, z);
      if CONSTEXPR (do_e) {
         e0tt += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0ttxx += cvt_to<vbuf_prec>(vxx);
         v0ttyx += cvt_to<vbuf_prec>(vyx);
         v0ttzx += cvt_to<vbuf_prec>(vzx);
         v0ttyy += cvt_to<vbuf_prec>(vyy);
         v0ttzy += cvt_to<vbuf_prec>(vzy);
         v0ttzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if CONSTEXPR (do_e and rc_a) {
      if (ntortor > 0)
         atomic_add(e0tt, ett, ithread);
   }
   if CONSTEXPR (do_v and rc_a) {
      if (ntortor > 0)
         atomic_add(v0ttxx, v0ttyx, v0ttzx, v0ttyy, v0ttzy, v0ttzz, vir_ett,
                    ithread);
   }


   // egeom position
   for (int i = ithread; i < npfix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom_position<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                            degx, degy, degz,

                            i, ipfix, kpfix, xpfix, ypfix, zpfix, pfix,

                            x, y, z, TINKER_IMAGE_ARGS);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   // egeom group
   for (int i = ithread; i < ngfix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom_group<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                         degx, degy, degz,

                         i, igfix, gfix,

                         x, y, z, mass, molec, igrp, kgrp, grpmass,
                         TINKER_IMAGE_ARGS);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   // egeom distance
   for (int i = ithread; i < ndfix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom_distance<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                            degx, degy, degz,

                            i, idfix, dfix,

                            x, y, z, molec, TINKER_IMAGE_ARGS);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   // egeom angle
   for (int i = ithread; i < nafix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom_angle<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                         degx, degy, degz,

                         i, iafix, afix, x, y, z);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   // egeom torsion
   for (int i = ithread; i < ntfix; i += stride) {
      real e, vxx, vyx, vzx, vyy, vzy, vzz;
      dk_geom_torsion<Ver>(e, vxx, vyx, vzx, vyy, vzy, vzz,

                           degx, degy, degz,

                           i, itfix, tfix, x, y, z);
      if CONSTEXPR (do_e) {
         e0g += cvt_to<ebuf_prec>(e);
      }
      if CONSTEXPR (do_v) {
         v0gxx += cvt_to<vbuf_prec>(vxx);
         v0gyx += cvt_to<vbuf_prec>(vyx);
         v0gzx += cvt_to<vbuf_prec>(vzx);
         v0gyy += cvt_to<vbuf_prec>(vyy);
         v0gzy += cvt_to<vbuf_prec>(vzy);
         v0gzz += cvt_to<vbuf_prec>(vzz);
      }
   }
   if (npfix + ngfix + ndfix + nafix + ntfix > 0) {
      if CONSTEXPR (do_e and rc_a) {
         atomic_add(e0g, eg, ithread);
      }
      if CONSTEXPR (do_v and rc_a) {
         atomic_add(v0gxx, v0gyx, v0gzx, v0gyy, v0gzy, v0gzz, vir_eg, ithread);
      }
   }


   // total energy and virial
   if CONSTEXPR (do_e and not rc_a) {
      ebuf_prec etl = 0;
      etl += e0b;   // ebond
      etl += e0a;   // eangle
      etl += e0ba;  // estrbnd
      etl += e0ub;  // eurey
      etl += e0opb; // eopbend
      etl += e0id;  // eimprop
      etl += e0it;  // eimptor
      etl += e0t;   // etors
      etl += e0pt;  // epitors
      etl += e0tt;  // etortor
      etl += e0g;   // egeom
      atomic_add(etl, ebuf, ithread);
   }
   if CONSTEXPR (do_v and not rc_a) {
      vbuf_prec vtlxx = 0, vtlyx = 0, vtlzx = 0;
      vbuf_prec vtlyy = 0, vtlzy = 0, vtlzz = 0;
      // ebond
      vtlxx += v0bxx, vtlyx += v0byx, vtlzx += v0bzx;
      vtlyy += v0byy, vtlzy += v0bzy, vtlzz += v0bzz;
      // eangle
      vtlxx += v0axx, vtlyx += v0ayx, vtlzx += v0azx;
      vtlyy += v0ayy, vtlzy += v0azy, vtlzz += v0azz;
      // estrbnd
      vtlxx += v0baxx, vtlyx += v0bayx, vtlzx += v0bazx;
      vtlyy += v0bayy, vtlzy += v0bazy, vtlzz += v0bazz;
      // eurey
      vtlxx += v0ubxx, vtlyx += v0ubyx, vtlzx += v0ubzx;
      vtlyy += v0ubyy, vtlzy += v0ubzy, vtlzz += v0ubzz;
      // eopbend
      vtlxx += v0opbxx, vtlyx += v0opbyx, vtlzx += v0opbzx;
      vtlyy += v0opbyy, vtlzy += v0opbzy, vtlzz += v0opbzz;
      // eimprop
      vtlxx += v0idxx, vtlyx += v0idyx, vtlzx += v0idzx;
      vtlyy += v0idyy, vtlzy += v0idzy, vtlzz += v0idzz;
      // eimptor
      vtlxx += v0itxx, vtlyx += v0ityx, vtlzx += v0itzx;
      vtlyy += v0ityy, vtlzy += v0itzy, vtlzz += v0itzz;
      // etors
      vtlxx += v0txx, vtlyx += v0tyx, vtlzx += v0tzx;
      vtlyy += v0tyy, vtlzy += v0tzy, vtlzz += v0tzz;
      // epitors
      vtlxx += v0ptxx, vtlyx += v0ptyx, vtlzx += v0ptzx;
      vtlyy += v0ptyy, vtlzy += v0ptzy, vtlzz += v0ptzz;
      // etortor
      vtlxx += v0ttxx, vtlyx += v0ttyx, vtlzx += v0ttzx;
      vtlyy += v0ttyy, vtlzy += v0ttzy, vtlzz += v0ttzz;
      // egeom
      vtlxx += v0gxx, vtlyx += v0gyx, vtlzx += v0gzx;
      vtlyy += v0gyy, vtlzy += v0gzy, vtlzz += v0gzz;
      atomic_add(vtlxx, vtlyx, vtlzx, vtlyy, vtlzy, vtlzz, vbuf, ithread);
   }
}


// clang-format off
#define EVALENCE_ARGS                                                          \
   /* ebond */ eb, vir_eb, debx, deby, debz, bndtyp, bndunit,                  \
   flag_bond ? nbond : 0, ibnd, bl, bk, cbnd, qbnd,                            \
   /* eangle */ ea, vir_ea, deax, deay, deaz, angtyp, angunit,                 \
   flag_angle ? nangle : 0, iang, anat, ak, afld, cang, qang, pang, sang,      \
   /* estrbnd */ eba, vir_eba, debax, debay, debaz, stbnunit,                  \
   flag_strbnd ? nstrbnd : 0, isb, sbk,                                        \
   /* eurey */ eub, vir_eub, deubx, deuby, deubz, ureyunit,                    \
   flag_urey ? nurey : 0, iury, uk, ul, cury, qury,                            \
   /* eopbend */ eopb, vir_eopb, deopbx, deopby, deopbz, opbtyp, opbunit,      \
   flag_opb ? nopbend : 0, iopb, opbk, copb, qopb, popb, sopb,                 \
   /* eimprop */ eid, vir_eid, deidx, deidy, deidz, idihunit,                  \
   flag_improp ? niprop : 0, iiprop, kprop, vprop,                             \
   /* eimptor */ eit, vir_eit, deitx, deity, deitz, itorunit,                  \
   flag_imptor ? nitors : 0, iitors, itors1, itors2, itors3,                   \
   /* etors */ et, vir_et, detx, dety, detz, torsunit,                         \
   flag_tors ? ntors : 0, itors, tors1, tors2, tors3, tors4, tors5, tors6,     \
   /* epitors */ ept, vir_ept, deptx, depty, deptz, ptorunit,                  \
   flag_pitors ? npitors : 0, ipit, kpit,                                      \
   /* estrtor */ ebt, vir_ebt, debtx, debty, debtz, storunit,                  \
   flag_strtor ? nstrtor : 0, ist, kst,                                        \
   /* eangtor */ eat, vir_eat, deatx, deaty, deatz, atorunit,                  \
   flag_angtor ? nangtor : 0, iat, kant,                                       \
   /* etortor */ ett, vir_ett, dettx, detty, dettz, ttorunit,                  \
   flag_tortor ? ntortor : 0, itt, ibitor, chkttor_ia_,                        \
   tnx, tny, ttx, tty, tbf, tbx, tby, tbxy,                                    \
   /* egeom */ eg, vir_eg, degx, degy, degz,                                   \
   flag_geom ? npfix : 0, ipfix, kpfix, xpfix, ypfix, zpfix, pfix,             \
   flag_geom ? ngfix : 0, igfix, gfix,                                         \
   flag_geom ? ndfix : 0, idfix, dfix,                                         \
   flag_geom ? nafix : 0, iafix, afix,                                         \
   flag_geom ? ntfix : 0, itfix, tfix,                                         \
   /* total */ eng_buf, vir_buf,                                               \
   /* other */ x, y, z, mass, molecule.molecule,                               \
   grp.igrp, grp.kgrp, grp.grpmass, TINKER_IMAGE_ARGS
// clang-format on
void evalence_cu2(int vers, bool flag_bond, bool flag_angle, bool flag_strbnd,
                  bool flag_urey, bool flag_opb, bool flag_improp,
                  bool flag_imptor, bool flag_tors, bool flag_pitors,
                  bool flag_strtor, bool flag_angtor, bool flag_tortor,
                  bool flag_geom)
{
   int ngrid = get_grid_size(BLOCK_DIM);
   if (rc_flag & calc::analyz) {
      if (vers == calc::v0 or vers == calc::v3)
         evalence_cu1<calc::V0, true>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v1)
         evalence_cu1<calc::V1, true>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v4)
         evalence_cu1<calc::V4, true>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v5)
         evalence_cu1<calc::V5, true>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v6)
         evalence_cu1<calc::V6, true>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
   } else {
      if (vers == calc::v0)
         evalence_cu1<calc::V0, false>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v1)
         evalence_cu1<calc::V1, false>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v3)
         assert(false);
      else if (vers == calc::v4)
         evalence_cu1<calc::V4, false>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v5)
         evalence_cu1<calc::V5, false>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
      else if (vers == calc::v6)
         evalence_cu1<calc::V6, false>
            <<<ngrid, BLOCK_DIM, 0, g::s0>>>(EVALENCE_ARGS);
   }
}
#undef EVALENCE_ARGS


void evalence_cu(int vers)
{
   bool rc_a = rc_flag & calc::analyz;
   bool do_e = vers & calc::energy;
   bool do_v = vers & calc::virial;
   bool do_g = vers & calc::grad;


   bool flag_bond = use_potent(bond_term);
   bool flag_angle = use_potent(angle_term);
   bool flag_strbnd = use_potent(strbnd_term);
   bool flag_urey = use_potent(urey_term);
   bool flag_opb = use_potent(opbend_term);
   bool flag_improp = use_potent(improp_term);
   bool flag_imptor = use_potent(imptors_term);
   bool flag_tors = use_potent(torsion_term);
   bool flag_pitors = use_potent(pitors_term);
   bool flag_strtor = use_potent(strtor_term);
   bool flag_angtor = use_potent(angtor_term);
   bool flag_tortor = use_potent(tortor_term);
   bool flag_geom = use_potent(geom_term);


   size_t bsize = buffer_size();
   if (rc_a and flag_bond) {
      host_zero(energy_eb, virial_eb);
      if (do_e)
         darray::zero(g::q0, bsize, eb);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eb);
      if (do_g)
         darray::zero(g::q0, n, debx, deby, debz);
   }
   if (rc_a and flag_angle) {
      host_zero(energy_ea, virial_ea);
      if (do_e)
         darray::zero(g::q0, bsize, ea);
      if (do_v)
         darray::zero(g::q0, bsize, vir_ea);
      if (do_g)
         darray::zero(g::q0, n, deax, deay, deaz);
   }
   if (rc_a and flag_strbnd) {
      host_zero(energy_eba, virial_eba);
      if (do_e)
         darray::zero(g::q0, bsize, eba);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eba);
      if (do_g)
         darray::zero(g::q0, n, debax, debay, debaz);
   }
   if (rc_a and flag_urey) {
      host_zero(energy_eub, virial_eub);
      if (do_e)
         darray::zero(g::q0, bsize, eub);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eub);
      if (do_g)
         darray::zero(g::q0, n, deubx, deuby, deubz);
   }
   if (rc_a and flag_opb) {
      host_zero(energy_eopb, virial_eopb);
      if (do_e)
         darray::zero(g::q0, bsize, eopb);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eopb);
      if (do_g)
         darray::zero(g::q0, n, deopbx, deopby, deopbz);
   }
   if (rc_a and flag_improp) {
      if (do_e)
         darray::zero(g::q0, bsize, eid);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eid);
      if (do_g)
         darray::zero(g::q0, n, deidx, deidy, deidz);
   }
   if (rc_a and flag_imptor) {
      if (do_e)
         darray::zero(g::q0, bsize, eit);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eit);
      if (do_g)
         darray::zero(g::q0, n, deitx, deity, deitz);
   }
   if (rc_a and flag_tors) {
      host_zero(energy_et, virial_et);
      if (do_e)
         darray::zero(g::q0, bsize, et);
      if (do_v)
         darray::zero(g::q0, bsize, vir_et);
      if (do_g)
         darray::zero(g::q0, n, detx, dety, detz);
   }
   if (rc_a and flag_pitors) {
      host_zero(energy_ept, virial_ept);
      if (do_e)
         darray::zero(g::q0, bsize, ept);
      if (do_v)
         darray::zero(g::q0, bsize, vir_ept);
      if (do_g)
         darray::zero(g::q0, n, deptx, depty, deptz);
   }
   if (rc_a and flag_strtor) {
      host_zero(energy_ebt, virial_ebt);
      if (do_e)
         darray::zero(g::q0, bsize, ebt);
      if (do_v)
         darray::zero(g::q0, bsize, vir_ebt);
      if (do_g)
         darray::zero(g::q0, n, debtx, debty, debtz);
   }
   if (rc_a and flag_angtor) {
      host_zero(energy_eat, virial_eat);
      if (do_e)
         darray::zero(g::q0, bsize, eat);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eat);
      if (do_g)
         darray::zero(g::q0, n, deatx, deaty, deatz);
   }
   if (rc_a and flag_tortor) {
      host_zero(energy_ett, virial_ett);
      if (do_e)
         darray::zero(g::q0, bsize, ett);
      if (do_v)
         darray::zero(g::q0, bsize, vir_ett);
      if (do_g)
         darray::zero(g::q0, n, dettx, detty, dettz);
   }
   if (rc_a and flag_geom) {
      host_zero(energy_eg, virial_eg);
      if (do_e)
         darray::zero(g::q0, bsize, eg);
      if (do_v)
         darray::zero(g::q0, bsize, vir_eg);
      if (do_g)
         darray::zero(g::q0, n, degx, degy, degz);
   }


   if (flag_bond or flag_angle or flag_strbnd or flag_urey or flag_opb or
       flag_improp or flag_imptor or flag_tors or flag_pitors or flag_strtor or
       flag_angtor or flag_tortor or flag_geom) {
      evalence_cu2(vers, flag_bond, flag_angle, flag_strbnd, flag_urey,
                   flag_opb, flag_improp, flag_imptor, flag_tors, flag_pitors,
                   flag_strtor, flag_angtor, flag_tortor, flag_geom);
   }


   if (rc_a and flag_bond) {
      if (do_e) {
         energy_eb = energy_reduce(eb);
         energy_valence += energy_eb;
      }
      if (do_v) {
         virial_reduce(virial_eb, vir_eb);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eb[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, debx, deby, debz);
   }
   if (rc_a and flag_angle) {
      if (do_e) {
         energy_ea = energy_reduce(ea);
         energy_valence += energy_ea;
      }
      if (do_v) {
         virial_reduce(virial_ea, vir_ea);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ea[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deax, deay, deaz);
   }
   if (rc_a and flag_strbnd) {
      if (do_e) {
         energy_eba = energy_reduce(eba);
         energy_valence += energy_eba;
      }
      if (do_v) {
         virial_reduce(virial_eba, vir_eba);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eba[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, debax, debay, debaz);
   }
   if (rc_a and flag_urey) {
      if (do_e) {
         energy_eub = energy_reduce(eub);
         energy_valence += energy_eub;
      }
      if (do_v) {
         virial_reduce(virial_eub, vir_eub);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eub[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deubx, deuby, deubz);
   }
   if (rc_a and flag_opb) {
      if (do_e) {
         energy_eopb = energy_reduce(eopb);
         energy_valence += energy_eopb;
      }
      if (do_v) {
         virial_reduce(virial_eopb, vir_eopb);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eopb[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deopbx, deopby, deopbz);
   }
   if (rc_a and flag_improp) {
      if (do_e) {
         energy_eid = energy_reduce(eid);
         energy_valence += energy_eid;
      }
      if (do_v) {
         virial_reduce(virial_eid, vir_eid);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eid[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deidx, deidy, deidz);
   }
   if (rc_a and flag_imptor) {
      if (do_e) {
         energy_eit = energy_reduce(eit);
         energy_valence += energy_eit;
      }
      if (do_v) {
         virial_reduce(virial_eit, vir_eit);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eit[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deitx, deity, deitz);
   }
   if (rc_a and flag_tors) {
      if (do_e) {
         energy_et = energy_reduce(et);
         energy_valence += energy_et;
      }
      if (do_v) {
         virial_reduce(virial_et, vir_et);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_et[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, detx, dety, detz);
   }
   if (rc_a and flag_pitors) {
      if (do_e) {
         energy_ept = energy_reduce(ept);
         energy_valence += energy_ept;
      }
      if (do_v) {
         virial_reduce(virial_ept, vir_ept);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ept[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deptx, depty, deptz);
   }
   if (rc_a and flag_strtor) {
      if (do_e) {
         energy_ebt = energy_reduce(ebt);
         energy_valence += energy_ebt;
      }
      if (do_v) {
         virial_reduce(virial_ebt, vir_ebt);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ebt[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, debtx, debty, debtz);
   }
   if (rc_a and flag_angtor) {
      if (do_e) {
         energy_eat = energy_reduce(eat);
         energy_valence += energy_eat;
      }
      if (do_v) {
         virial_reduce(virial_eat, vir_eat);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eat[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, deatx, deaty, deatz);
   }
   if (rc_a and flag_tortor) {
      if (do_e) {
         energy_ett = energy_reduce(ett);
         energy_valence += energy_ett;
      }
      if (do_v) {
         virial_reduce(virial_ett, vir_ett);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_ett[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, dettx, detty, dettz);
   }
   if (rc_a and flag_geom) {
      if (do_e) {
         energy_eg = energy_reduce(eg);
         energy_valence += energy_eg;
      }
      if (do_v) {
         virial_reduce(virial_eg, vir_eg);
         for (int iv = 0; iv < 9; ++iv)
            virial_valence[iv] += virial_eg[iv];
      }
      if (do_g)
         sum_gradient(gx, gy, gz, degx, degy, degz);
   }
}
}
